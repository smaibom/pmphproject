#include "hip/hip_runtime.h"
#include "ProjHelperFun.cu.h"
#include "Constants.h"
#include "TridagPar.h"

#define BLOCK_SIZE 32



__global__ void updateParamsKernel(const unsigned g, const REAL alpha, 
                                   const REAL beta, const REAL nu, REAL* myVarX,
                                   REAL* myVarY, REAL* myY, REAL* myX, 
                                   REAL* myTimeline,const int numY, const int numM){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    myVarX[z*numM+i*numY+j] = exp(2.0*(beta*log(myX[i])+myY[j]-0.5*nu*nu*myTimeline[g]));
    myVarY[z* numM + i * numY + j] =exp(2.0*(  alpha*log(myX[i])+myY[j]- 0.5*nu*nu*myTimeline[g] ));
}

void updateParams(const unsigned g, const REAL alpha, const REAL beta, 
                  const REAL nu, PrivGlobs& globs, const int outer)
{
  int numX = globs.numX;
  int numY = globs.numY;
  int numT = globs.numT;
  int numM = numX*numY;

  //Device memory
  hipMemcpy(globs.dmyY, globs.myY, numY * sizeof(REAL), hipMemcpyHostToDevice);
  hipMemcpy(globs.dmyTimeline, globs.myTimeline, numT * sizeof(REAL), hipMemcpyHostToDevice);
  hipMemcpy(globs.dmyX, globs.myX, numX * sizeof(REAL), hipMemcpyHostToDevice);

  dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 numBlocks(numX / BLOCK_SIZE, numY / BLOCK_SIZE, outer);
  updateParamsKernel<<< numBlocks, threadsPerBlock >>> (g, alpha, beta, nu, globs.dmyVarX, 
                                                        globs.dmyVarY, globs.dmyY,globs.dmyX,globs.dmyTimeline,numY,numX*numY);

  hipMemcpy(globs.myVarX, globs.dmyVarX, outer * numX * numY * sizeof(REAL), hipMemcpyDeviceToHost);
  hipMemcpy(globs.myVarY, globs.dmyVarY, outer * numX * numY * sizeof(REAL), hipMemcpyDeviceToHost);
  REAL* myVarXNew = (REAL*) malloc(sizeof(REAL) * globs.numX * globs.numY * outer);
  transpose(globs.myVarX,myVarXNew,globs.numX,globs.numY,outer);
  free(globs.myVarX);
  globs.myVarX = myVarXNew;
}

__global__ void setPayoffKernel(REAL* myX, REAL*   myResult, unsigned int numX, unsigned int numY) {
    int i = BLOCK_SIZE * blockIdx.x + threadIdx.x;
    int j = BLOCK_SIZE * blockIdx.y + threadIdx.y;
    int o = blockIdx.z;
  
    myResult[o * (numY * numX) + i * numY + j] = 
      max(myX[i] - o * 0.001, (REAL)0.0);
}

void setPayoff_cuda(PrivGlobs& globs, unsigned int outer)
{ 
    REAL* myResult_d;
  hipMemcpy(globs.dmyX, globs.myX, globs.numX*sizeof(REAL ), hipMemcpyHostToDevice);
  
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks(globs.numX / BLOCK_SIZE, globs.numY / BLOCK_SIZE, outer);
  
  //kernel
    setPayoffKernel<<<numBlocks, threadsPerBlock>>>(globs.dmyX, globs.dmyResult, globs.numX, globs.numY);

  hipMemcpy(globs.myResult, globs.dmyResult, outer*globs.numX*globs.numY*sizeof(REAL), hipMemcpyDeviceToHost);
}

//All arrays are size [n]
inline void tridag(REAL* a,REAL* b,REAL* c,const REAL* r,const int n, 
  REAL* u, REAL* uu)
{
    int    i, offset;
    REAL   beta;

    u[0]  = r[0];
    uu[0] = b[0];

    for(i=1; i<n; i++) 
    {
        beta  = a[i] / uu[i-1];

        uu[i] = b[i] - beta*c[i-1];
        u[i]  = r[i] - beta*u[i-1];
    }

#if 1
    // X) this is a backward recurrence
    u[n-1] = u[n-1] / uu[n-1];
    for(i=n-2; i>=0; i--) 
    {
        u[i] = (u[i] - c[i]*u[i+1]) / uu[i];
    }
#else
    // Hint: X) can be written smth like (once you make a non-constant)
    for(i=0; i<n; i++) a[i] =  u[n-1-i];
    a[0] = a[0] / uu[n-1];
    for(i=1; i<n; i++) a[i] = (a[i] - c[n-1-i]*a[i-1]) / uu[n-1-i];
    for(i=0; i<n; i++) u[i] = a[n-1-i];
#endif
}


__global__ void rollback_implicit_y (REAL* y, REAL*  u, REAL* v, REAL dtInv, int numX, int numY) {
  int j = BLOCK_SIZE * blockIdx.x + threadIdx.x;
  int i = BLOCK_SIZE * blockIdx.y + threadIdx.y;
  int o = blockIdx.z;

  y[o * numX * numY + i * numY + j] = dtInv*u[o * numX * numY + j * numX + i]
    - 0.5*v[o * numX * numY + i * numY + j];

}


__global__ void rollback_x(REAL* ax, REAL* bx, REAL* cx, REAL* u, REAL* myVarX, REAL* myDxx, REAL* myResult,
                           REAL dtInv, int numX, int numY) {

  int i = BLOCK_SIZE * blockIdx.x + threadIdx.x;
  int j = BLOCK_SIZE * blockIdx.y + threadIdx.y;
  int o = blockIdx.z;

  int numM = numY * numX;

  ax[o * numX * numY + j * numX + i] = -0.5*(0.5*myVarX[o * numM + j * numX + i]*myDxx[i * 4 + 0]);

  bx[o * numX * numY + j * numX + i] =
    dtInv - 0.5*(0.5*myVarX[o * numM + j * numX + i]
                 *myDxx[i * 4 + 1]);
  cx[o * numX * numY + j * numX + i] =
    -0.5*(0.5*myVarX[o * numM + j * numX + i]
          *myDxx[i * 4 + 2]);
  //  explicit x
  u[o * numX * numY + j * numX + i] =
    dtInv*myResult[o * numM + i * numY + j];

  if(i > 0) {
      u[o * numX * numY + j * numX + i] +=
        0.5*(0.5*myVarX[o * numM + + j * numX + i]
             *myDxx[i * 4 + 0])
        *myResult[o * numM + (i-1) * numY + j];
    }

  u[o * numX * numY + j * numX + i]  +=
    0.5*(0.5*myVarX[o * numM + + j * numX + i]*myDxx[i * 4 + 1])
    *myResult[o * numM + i * numY + j];

  if(i < numX-1) {
      u[o * numX * numY + j * numX + i] +=
        0.5*(0.5*myVarX[o * numM + + j * numX + i]
             *myDxx[i * 4 + 2])
        *myResult[o * numM + (i+1) * numY + j];
    }
}


__global__ void rollback_y(REAL* ay, REAL* by, REAL* cy, REAL* u, REAL* v, REAL* myVarY, REAL* myDyy, REAL* myResult,
                           REAL dtInv, int numX, int numY) {
  int j = BLOCK_SIZE * blockIdx.x + threadIdx.x;
  int i = BLOCK_SIZE * blockIdx.y + threadIdx.y;
  int o = blockIdx.z;
  int numM = numX * numY;

  v[o * numX * numY + i * numY + j] = 0.0;

  if(j > 0) 
    {
      v[o * numX * numY + i * numY + j] +=  (0.5*myVarY[o * numM + i * numY + j]*myDyy[j * 4 + 0])
  *myResult[o * numM + i * numY + j-1];
    }

  v[o * numX * numY + i * numY + j] += (0.5*myVarY[o * numM + i * numY + j]
          *myDyy[j * 4 + 1])
    * myResult[o * numM + i  * numY + j];

  if(j < numY-1) 
    {
      v[o * numX * numY + i * numY + j] +=  
  (0.5*myVarY[o * numM + i * numY + j]
   *myDyy[j * 4 + 2])
  *myResult[o * numM + i * numY + j+1];
    }

  u[o * numX * numY + j * numX + i] += v[o * numX * numY + i * numY + j];

  // Implicit y

  ay[o * numX * numY + i * numY + j] =
    -0.5*(0.5*myVarY[o * numM + i * numY + j]
    *myDyy[j * 4 + 0]);

  by[o * numX * numY + i * numY + j] = 
    dtInv - 0.5*(0.5*myVarY[o * numM + i * numY + j]
     *myDyy[j * 4 + 1]);

  cy[o * numX * numY + i * numY + j] =
    -0.5*(0.5*myVarY[o * numM + i * numY + j]
    *myDyy[j * 4 + 2]);
}



void
rollback( const unsigned g, PrivGlobs& globs, int outer, const int& numX, 
          const int& numY) 
{
  unsigned numZ = max(numX,numY);
  unsigned numM = numX * numY;

  REAL* u = (REAL*) malloc(sizeof(REAL) * outer * numY * numX);   // [outer][numY][numX]
  REAL* v = (REAL*) malloc(sizeof(REAL) * outer * numX * numY);   // [outer][numX][numY]
  REAL* ax = (REAL*) malloc(sizeof(REAL) * outer * numX * numY); // [outer][numY][numX]
  REAL* bx = (REAL*) malloc(sizeof(REAL) * outer * numX * numY); // [outer][numY][numX]
  REAL* cx = (REAL*) malloc(sizeof(REAL) * outer * numX * numY); // [outer][numY][numX]
  REAL* ay = (REAL*) malloc(sizeof(REAL) * outer * numX * numY); // [outer][numX][numY]
  REAL* by = (REAL*) malloc(sizeof(REAL) * outer * numX * numY); // [outer][numX][numY]
  REAL* cy = (REAL*) malloc(sizeof(REAL) * outer * numX * numY); // [outer][numX][numY]
  REAL* y = (REAL*) malloc(sizeof(REAL) * outer * numX * numY); // [outer][numZ][numZ]
  REAL* yy = (REAL*) malloc(sizeof(REAL)*outer*numZ); // [outer][numZ]

  //Device memory

  hipMemcpy(globs.du, u, outer * numX * numY * sizeof(REAL), hipMemcpyHostToDevice);
  hipMemcpy(globs.dmyResult, globs.myResult, outer * numX * numY * sizeof(REAL), hipMemcpyHostToDevice);
  hipMemcpy(globs.dmyVarX, globs.myVarX, outer * numX * numY * sizeof(REAL), hipMemcpyHostToDevice);
  hipMemcpy(globs.dmyDxx, globs.myDxx, outer * numX * 4 * sizeof(REAL), hipMemcpyHostToDevice);

  dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 numBlocks(numX / BLOCK_SIZE, numY / BLOCK_SIZE, outer);

  REAL dtInv = 1.0/(globs.myTimeline[g+1]-globs.myTimeline[g]);
  rollback_x<<< numBlocks, threadsPerBlock >>> (globs.dax, globs.dbx, globs.dcx, globs.du, globs.dmyVarX, globs.dmyDxx, globs.dmyResult,
            dtInv, numX, numY);

  hipMemcpy(ax, globs.dax, outer * numX * numY * sizeof(REAL), hipMemcpyDeviceToHost);
  hipMemcpy(bx, globs.dbx, outer * numX * numY * sizeof(REAL), hipMemcpyDeviceToHost);
  hipMemcpy(cx, globs.dcx, outer * numX * numY * sizeof(REAL), hipMemcpyDeviceToHost);


  REAL* dv,* dmyVarY,* dmyDyy,* day,* dby,* dcy;

  hipMalloc((void**)&day, outer * numX * numY * sizeof(REAL));
  hipMalloc((void**)&dby, outer * numX * numY * sizeof(REAL));
  hipMalloc((void**)&dcy, outer * numX * numY * sizeof(REAL));
  hipMalloc((void**)&dv, outer * numX * numY * sizeof(REAL));

  hipMalloc((void**)&dmyVarY, outer * numX * numY * sizeof(REAL));
  hipMalloc((void**)&dmyDyy, outer * numY * 4 * sizeof(REAL));


  hipMemcpy(dmyVarY, globs.myVarY, outer * numX * numY * sizeof(REAL), hipMemcpyHostToDevice);
  hipMemcpy(dmyDyy, globs.myDyy, outer * numX * 4 * sizeof(REAL), hipMemcpyHostToDevice);


  numBlocks.x = numY / BLOCK_SIZE;
  numBlocks.y = numX / BLOCK_SIZE;

  rollback_y<<< numBlocks, threadsPerBlock >>> (day, dby, dcy, globs.du, dv, dmyVarY, dmyDyy, globs.dmyResult,
            dtInv, numX, numY);

  hipMemcpy(ay, day, outer * numX * numY * sizeof(REAL), hipMemcpyDeviceToHost);
  hipMemcpy(by, dby, outer * numX * numY * sizeof(REAL), hipMemcpyDeviceToHost);
  hipMemcpy(cy, dcy, outer * numX * numY * sizeof(REAL), hipMemcpyDeviceToHost);
  hipMemcpy(v, dv, outer * numX * numY * sizeof(REAL), hipMemcpyDeviceToHost);
  hipMemcpy(u, globs.du, outer * numX * numY * sizeof(REAL), hipMemcpyDeviceToHost);



#pragma omp parallel for default(shared) schedule(static) if(outer>8)
  for(int o = 0; o < outer; o++) 
  {
    for(int j=0;j<numY;j++) 
      {
      // here yy should have size [numX]
      tridagPar(&ax[o * numX * numY + j * numX],&bx[o * numX * numY + j * numX],
                &cx[o * numX * numY + j * numX], &u[o * numX * numY + j * numX], 
                numX, &u[o * numX * numY + numX * j], &yy[o*numZ]);
    }
  }


  REAL* dy;

  hipMalloc((void**)&dy, outer * numY * numX * sizeof(REAL));


  hipMemcpy(globs.du, u, outer * numX * numY * sizeof(REAL), hipMemcpyHostToDevice);
  rollback_implicit_y<<< numBlocks, threadsPerBlock >>> (dy, globs.du, dv,
            dtInv, numX, numY);
  hipMemcpy(y, dy, outer * numX * numY * sizeof(REAL), hipMemcpyDeviceToHost);

  // for(int o = 0; o < outer; o++)
  // {
  //   REAL dtInv = 1.0/(globs.myTimeline[g+1]-globs.myTimeline[g]);
  //   for(int i=0;i<numX;i++)
  //   {
  //     for(int j=0;j<numY;j++)
  //     {  // here a, b, c should have size [numY]
  //       y[o * numX * numY + i * numY + j] =
  //         dtInv*u[o * numX * numY + j * numX + i]
  //          -0.5*v[o * numX * numY + i * numY + j];
  //     }
  //   }
  // }

#pragma omp parallel for default(shared) schedule(static) if(outer>8)
  for(int o = 0; o < outer; o++)
  {
    for(int i=0;i<numX;i++)
    {
      // here yy should have size [numY]
      tridagPar(&ay[o * numX * numY + i * numY],&by[o * numX * numY + i * numY],
                &cy[o * numX * numY + i * numY],&y[o * numX * numY + i * numY],
                numY,&globs.myResult[o * numM + i * numY],&yy[o*numZ]);
    }
  }


  /* Free Memory */
  hipFree(dv);
  hipFree(dmyVarY);
  hipFree(dmyDyy);
  hipFree(day);
  hipFree(dby);
  hipFree(dcy);
  
  
  free(u);
  free(ax);
  free(ay);
  free(bx);
  free(by);
  free(cx);
  free(cy);
  free(yy);
  free(y);
}



void   run_OrigCPU(const unsigned int& outer,const unsigned int& numX,
                   const unsigned int& numY,const unsigned int& numT,
                   const REAL& s0,const REAL& t,const REAL& alpha,
                   const REAL& nu,const REAL& beta,REAL* res) // [outer] RESULT
{
  PrivGlobs globals(numX, numY, numT, outer);
  initGrid(s0,alpha,nu,t, numX, numY, numT, globals);
  initOperator(globals.myX, globals.numX, globals.myDxx);
  initOperator(globals.myY, globals.numY, globals.myDyy);


  setPayoff_cuda(globals, outer);



  for(int g = numT-2;g>=0;--g)
    {
      updateParams(g,alpha,beta,nu,globals, outer);
      rollback(g, globals, outer, numX, numY);
    }
  for (unsigned int i = 0; i < outer; i++) 
  {
    res[i] = globals.myResult[i * globals.numM + globals.myXindex * numY + globals.myYindex];
  }
}

//#endif // PROJ_CORE_ORIG
